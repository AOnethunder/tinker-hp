#include "hip/hip_runtime.h"
#include "utils.h"

#if (defined(SINGLE)||defined(MIXED))
#   define cuPOTRF_buffSize hipsolverDnSpotrf_bufferSize
#   if (CUDART_VERSION>10010)
#     define cuGESV_buffSize hipsolverDnSSgesv_bufferSize
#     define cuGESV           hipsolverDnSSgesv
#   else
#     define cuGETRF_buffsize hipsolverDnSgetrf_bufferSize
#     define cuGETRF          hipsolverDnSgetrf
#     define cuGETRS          hipsolverDnSgetrs
#   endif
#   define cuPOTRF          hipsolverDnSpotrf
#   define cuPOTRS          hipsolverDnSpotrs
#else
#   define cuPOTRF_buffSize hipsolverDnDpotrf_bufferSize
#   if (CUDART_VERSION>10010)
#     define cuGESV_buffSize  hipsolverDnDDgesv_bufferSize
#     define cuGESV           hipsolverDnDDgesv
#   else
#     define cuGETRF_buffsize hipsolverDnDgetrf_bufferSize
#     define cuGETRF         hipsolverDnDgetrf
#     define cuGETRS         hipsolverDnDgetrs
#   endif
#   define cuPOTRF          hipsolverDnDpotrf
#   define cuPOTRS          hipsolverDnDpotrs
#endif

extern const int rank;
extern const int tinkerdebug;

/* ---------
   Cu Solver global environnement
   ---------
*/
hipsolverHandle_t cuCholHandle = NULL;
const hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
real* d_workSpace=NULL;
size_t s_workSpaceSize;
int info;
int* d_info;

__global__ void CheckcuSolverInfo (int* d_info, int line, int rank) {
  if (*d_info != 0) printf (" Error info %d with cuSolver in " __FILE__ " :%d \n", *d_info, line);
}

EXTERN_C_BEG

void initcuSolverHandle(hipStream_t stream){
   gpuErrchkSolver( hipsolverDnCreate(&cuCholHandle) )

   gpuErrchkSolver( hipsolverSetStream(cuCholHandle, stream) )

   gpuErrchk( hipMalloc (&d_info, sizeof(int)) )

   if (rank==0) printf ("\n *** Using CuSolver Library ***\n\n" );
}

/* ----------------
   Reallocation procedure based on MOD_utilgpu.f reallocate_acc
   ---------------- */
void device_reallocate(void** array, const size_t bytesSize, size_t& PrevSize){
   if ( !(*array) ){
      gpuErrchk( hipMalloc(array,bytesSize) )
      PrevSize = bytesSize;
      //printf(" device_reallocate size %lu \n", PrevSize);
   }
   else {
      if (bytesSize > PrevSize) {
         gpuErrchk( hipFree( *array ) )
         gpuErrchk( hipMalloc(array,bytesSize) )
         PrevSize = bytesSize;
         /*printf(" device_reallocate size %lu \n", PrevSize);*/
      }
   }
}

void cuPOTRF_Wrapper(const int n, real* A, const int lda, hipStream_t stream){
   int Lwork=0;
   hipsolverStatus_t status1;

   gpuErrchk( cuPOTRF_buffSize(cuCholHandle, uplo, n, A, lda, &Lwork) )

   device_reallocate((void**)&d_workSpace, (size_t)Lwork*sizeof(real), s_workSpaceSize);

   status1 = cuPOTRF(cuCholHandle,uplo, n, A, lda, d_workSpace, Lwork, d_info);
   if (status1!=HIPSOLVER_STATUS_SUCCESS) printf( "Cholesky Factorisation on device failed with Error %d \n",status1 );

   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }

}

void cuPOTRS_Wrapper(const int n, real* A, const int lda, real* B, const int ldb, hipStream_t stream){
   hipsolverStatus_t status1;

   status1 = cuPOTRS(cuCholHandle, uplo, n, 1, A, lda, B, ldb, d_info);
   if (status1!=HIPSOLVER_STATUS_SUCCESS) printf( "Error %d solving Linear system \n",status1);

   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }
}

void cuGESV_Wrapper(const int n, const int nrhs, real* A, const int lda, int* Ipiv, real* B, const int ldb, hipStream_t stream){
#if (CUDART_VERSION>10010)
   size_t lwork_bytes=0;
   int iter;
   gpuErrchkSolver( cuGESV_buffSize(cuCholHandle, n, nrhs, A, lda, Ipiv, B, ldb, B, ldb, d_workSpace, &lwork_bytes) )
   device_reallocate((void**)&d_workSpace, (size_t)lwork_bytes, s_workSpaceSize);

   gpuErrchkSolver( cuGESV(cuCholHandle, n, nrhs, A, lda, Ipiv, B, ldb, B, ldb, d_workSpace, lwork_bytes, &iter, d_info) )
#else
   int Lwork=0;
   gpuErrchkSolver( cuGETRF_buffsize(cuCholHandle, n, n, A, lda, &Lwork) )
   //printf(" LU solve n %d nrhs %d lda %d ldb %d Lwork %d\n",n,nrhs,lda,ldb, Lwork);
   device_reallocate((void**)&d_workSpace, (size_t)Lwork*sizeof(real), s_workSpaceSize);

   gpuErrchkSolver( cuGETRF(cuCholHandle, n, n, A, lda, d_workSpace, Ipiv, d_info) )
   gpuErrchkSolver( cuGETRS(cuCholHandle, HIPBLAS_OP_N, n, 1, A, lda, Ipiv, B, ldb, d_info) )
#endif
   if (tinkerdebug) {
      CheckcuSolverInfo<<<1,1,0,stream>>>(d_info, __LINE__, rank);
      gpuErrchk( hipGetLastError() )
   }
}

void destroycuSolverHandle(){
   gpuErrchkSolver( hipsolverDnDestroy(cuCholHandle) )
}
EXTERN_C_END
